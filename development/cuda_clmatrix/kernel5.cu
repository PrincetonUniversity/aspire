
#include <hip/hip_runtime.h>
// Compute the correlations between all rays of two projections, without 
// taking shifts into account.
//
// This function has no test code, as this kernel was replaced by it 
// successors, such as kernel6.cu and on.
//
// Yoel Shkolnisky, May 2022.

// Add two complex numbers.
// Given two complex numbers a and b, reuturb a+b.     
__device__ __forceinline__ float2 ComplexAdd(float2 a, float2 b)
{
    float2 c;

    c.x = a.x + b.x;
    c.y = a.y + b.y;

    return c;
}

// Multiply two complex numbers.
// Given two complex numbers a and b, reuturb a*b.     
__device__ __forceinline__ float2 ComplexMul(float2 a, float2 b)
{
    float2 c;

    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;

    return c;
}

// Multiply two complex numbers with conjugation.
// Given two complex numbers a and b, reuturb conj(a)*b.
__device__ __forceinline__ float2 ComplexMulConj(float2 a, float2 b)
{
    float2 c;

    c.x = a.x * b.x + a.y * b.y;
    c.y = a.x * b.y - a.y * b.x;

    return c;
}

// Multiply two matrices
// Given A of size nxp, B of size pxm, and a target matrix C of size nxm, 
// compute C=A*B.
__device__ __forceinline__ void ComplexMatMul(const float2* A, const float2* B, 
        float2* C, const int n, const int m, const int p){
// A of size nxp, B of size pxm, C of size nxm.
    float2 ab;
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < m; ++j)
            for (int k = 0; k < p; ++k){
                ab = ComplexMul(A[i+n*k],B[k+p*j]);
                C[i+n*j] = ComplexAdd(C[i+n*j],ab);
            }
}

// Multiply two matrices while conjugating the first matrix.
// Given A of size pxn, B of size pxm, and a target matrix C of size nxm, 
// compute C=A'*B. (prime is conjugate transpose).
__device__ __forceinline__ void ComplexMatMulConj(const float2* A, const float2* B, 
        float2* C, const int n, const int m, const int p){
// A of size pxn, B of size pxm, C of size nxm.
    float2 ab;
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < m; ++j)
            for (int k = 0; k < p; ++k){
                ab = ComplexMulConj(A[k+p*i],B[k+p*j]);
                C[i+n*j] = ComplexAdd(C[i+n*j],ab);
            }
}

// Multiply two matrices while transposing (without conjugation) the first matrix.
// Given A of size pxn, B of size pxm, and a target matrix C of size nxm, 
// compute C=A'*B. (prime is conjugate transpose).
__device__ __forceinline__ void ComplexMatMulTrans(const float2* A, const float2* B, 
        float2* C, const int n, const int m, const int p){
// A of size pxn, B of size pxm, C of size nxm.
    float2 ab;
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < m; ++j)
            for (int k = 0; k < p; ++k){
                ab = ComplexMul(A[k+p*i],B[k+p*j]);
                C[i+n*j] = ComplexAdd(C[i+n*j],ab);
            }
}


// Take the real part of a matrix.
// A is complex with n elements. reA is float with n elements.
// Note that if A and reA are matrices, then n should be the total number
// of elements in the matrices.        
__device__ __forceinline__ void real(const float2* A, float* reA, const int n){
    for (int i=0; i<n; i++)
        reA[i] = A[i].x;
}

// Pointwise multiplication of a matrix by vector. The vector is expanded
// to all columns of the matrix.
// A is of size nxm. V is of size n. AV is of size nxm and is equal to 
// AV=bsxfun(@times,A,V);        
__device__ __forceinline__ void MatVecPointwise(const float2* A, const float2* V, 
        float2* AV, const int n, const int m){
    for (int j=0; j<m; j++){
        for (int i=0; i<n; i++){
            AV[i+j*n] = ComplexMul(A[i+j*n],V[i]);
        }
    }
}

// Find the maximum of a float array A with n elements.
// Returns the maximal value and the index of the maximum.
// If the same maximal value occurs several times, the first occurrence is 
// returned.         
// Note that the output index variable is of type single, so that all types
// in the calling function are of type single (to avoid mixing int32).        
__device__ __forceinline__ void max(const float* A, const int n, float* val, float* idx ){
    *val=A[0];
    *idx=0.0f;

    for (int i=1; i<n; i++){
        if (A[i]>*val){
            *val=A[i];
            *idx=(float) i;
        }
    }
}

__global__ void commonline(const float2* P1, const float2* P2, 
        const int n_r, const int n_theta, float *C1, float* C2, 
        float* cij, float* cji, float* corrstack){
    
    float ab;
    float maxcorr = -2.0; // Maximal correlation observed thus far.
    float cl12,cl21;        // The common line index at which the maxiaml 
            // correlation was obtained. These are floats so it will be 
            // easier to return to the calling host.
    float corr; // Correlation of the currently inpsected pair of lines.
    
// C1=2*real(P1_shifted'*P2);
// Essentially implemeting  
//        2.*real(ComplexMatMulConj(P1, P2, C2tmp, n_theta,n_theta,n_r));
// without allocating an addition array        
         
    for (int i = 0; i < n_theta; ++i){
        for (int j = 0; j < n_theta; ++j){
            corr = 0.0;
            for (int k = 0; k < n_r; ++k){
                // We only compute the real part of the product
                ab = P1[k+n_r*i].x * P2[k+n_r*j].x + P1[k+n_r*i].y * P2[k+n_r*j].y;
                corr += ab;
            }
            corr *= 2.0;
            C1[i+n_theta*j] = corr;

            if (corr>maxcorr){ // Current pair of lines is a better candidate for the common line
                    maxcorr = corr;
                    cl12 = (float) i;
                    cl21 = (float) j;
            }
        }
    }

//  C2=2*real(P1_shifted_flipped'*P2);
// Essentially implemeting  
//        2.*real(ComplexMatMulTrans(P1, P2, C2tmp, n_theta,n_theta,n_r));
// without allocating an addition array        
    for (int i = 0; i < n_theta; ++i){
        for (int j = 0; j < n_theta; ++j){
            corr = 0.0;
            for (int k = 0; k < n_r; ++k){
                // We only compute the real part of the product
                ab = P1[k+n_r*i].x * P2[k+n_r*j].x - P1[k+n_r*i].y * P2[k+n_r*j].y;
                corr += ab;
            }
            corr *= 2.0;
            C2[i+n_theta*j] = corr;

            if (corr>maxcorr){ // Current pair of lines is a better candidate for the common line
                    maxcorr = corr;
                    cl12 = (float) i;
                    cl21 = (float) j+n_theta;
            }
        }
    }        
    // Return results
    *cij = cl12 + 1.0;  // Host indexing is 1-based.
    *cji = cl21 + 1.0;
    *corrstack  = maxcorr;
}
