
#include <hip/hip_runtime.h>
// Like kernel7.cu, but instead of taking as input the indices of the 
// processed pair of images, take as input all stack of (Fourier 
// transformed) proejctions, and compute the index of the images to process
// from the grid paramters.
// No test code for this kernel, as the relevant test function has been 
// adapted to used kernel9.m instead.
//
// Yoel Shkolnisky, May 2022.

#define PI_F 3.141592654f

// Add two complex numbers.
// Given two complex numbers a and b, reuturb a+b.     
__device__ __forceinline__ float2 ComplexAdd(float2 a, float2 b)
{
    float2 c;

    c.x = a.x + b.x;
    c.y = a.y + b.y;

    return c;
}

// Multiply two complex numbers.
// Given two complex numbers a and b, reuturb a*b.     
__device__ __forceinline__ float2 ComplexMul(float2 a, float2 b)
{
    float2 c;

    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;

    return c;
}

// Multiply two complex numbers with conjugation.
// Given two complex numbers a and b, reuturb conj(a)*b.
__device__ __forceinline__ float2 ComplexMulConj(float2 a, float2 b)
{
    float2 c;

    c.x = a.x * b.x + a.y * b.y;
    c.y = a.x * b.y - a.y * b.x;

    return c;
}

// Exponential of a complex number.
__device__ __forceinline__ float2 ComplexExp(float2 z)

{
    float2 res;
    float t = expf (z.x);
    sincosf (z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
    return res;
}

// Multiply two matrices
// Given A of size nxp, B of size pxm, and a target matrix C of size nxm, 
// compute C=A*B.
__device__ __forceinline__ void ComplexMatMul(const float2* A, const float2* B, 
        float2* C, const int n, const int m, const int p){
// A of size nxp, B of size pxm, C of size nxm.
    float2 ab;
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < m; ++j)
            for (int k = 0; k < p; ++k){
                ab = ComplexMul(A[i+n*k],B[k+p*j]);
                C[i+n*j] = ComplexAdd(C[i+n*j],ab);
            }
}

// Multiply two matrices while conjugating the first matrix.
// Given A of size pxn, B of size pxm, and a target matrix C of size nxm, 
// compute C=A'*B. (prime is conjugate transpose).
__device__ __forceinline__ void ComplexMatMulConj(const float2* A, const float2* B, 
        float2* C, const int n, const int m, const int p){
// A of size pxn, B of size pxm, C of size nxm.
    float2 ab;
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < m; ++j)
            for (int k = 0; k < p; ++k){
                ab = ComplexMulConj(A[k+p*i],B[k+p*j]);
                C[i+n*j] = ComplexAdd(C[i+n*j],ab);
            }
}

// Multiply two matrices while transposing (without conjugation) the first matrix.
// Given A of size pxn, B of size pxm, and a target matrix C of size nxm, 
// compute C=A'*B. (prime is conjugate transpose).
__device__ __forceinline__ void ComplexMatMulTrans(const float2* A, const float2* B, 
        float2* C, const int n, const int m, const int p){
// A of size pxn, B of size pxm, C of size nxm.
    float2 ab;
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < m; ++j)
            for (int k = 0; k < p; ++k){
                ab = ComplexMul(A[k+p*i],B[k+p*j]);
                C[i+n*j] = ComplexAdd(C[i+n*j],ab);
            }
}

// Take the real part of a matrix.
// A is complex with n elements. reA is float with n elements.
// Note that if A and reA are matrices, then n should be the total number
// of elements in the matrices.        
__device__ __forceinline__ void real(const float2* A, float* reA, const int n){
    for (int i=0; i<n; i++)
        reA[i] = A[i].x;
}

// Pointwise multiplication of a matrix by vector. The vector is expanded
// to all columns of the matrix.
// A is of size nxm. V is of size n. AV is of size nxm and is equal to 
// AV=bsxfun(@times,A,V);        
__device__ __forceinline__ void MatVecPointwise(const float2* A, const float2* V, 
        float2* AV, const int n, const int m){
    for (int j=0; j<m; j++){
        for (int i=0; i<n; i++){
            AV[i+j*n] = ComplexMul(A[i+j*n],V[i]);
        }
    }
}

// Find the maximum of a float array A with n elements.
// Returns the maximal value and the index of the maximum.
// If the same maximal value occurs several times, the first occurrence is 
// returned.         
// Note that the output index variable is of type single, so that all types
// in the calling function are of type single (to avoid mixing int32).        
__device__ __forceinline__ void max(const float* A, const int n, float* val, float* idx ){
    *val=A[0];
    *idx=0.0f;

    for (int i=1; i<n; i++){
        if (A[i]>*val){
            *val=A[i];
            *idx=(float) i;
        }
    }
}

__global__ void commonline(const float2* P, const int n_projs,
        const int n_r, const int n_theta, const float max_shift, 
        const float shift_step, float* clstack, float* corrstack){
    
    int k1 =  blockIdx.x * blockDim.x + threadIdx.x;
    if (k1 >= n_projs) return;

    int k2 = blockIdx.y * blockDim.y + threadIdx.y;
    if (k2 >= n_projs) return;

    if (k1>=k2) return;

    const float2 *P1=P+k1*n_r*n_theta;
    const float2 *P2=P+k2*n_r*n_theta;
            
            
    float ab;
    float maxcorr = -2.0; // Maximal correlation observed thus far.
    float cl12,cl21;        // The common line index at which the maxiaml 
            // correlation was obtained. These are floats so it will be 
            // easier to return to the calling host.
    float corr; // Correlation of the currently inpsected pair of lines.
             
    float n_shifts = ceilf(2.0*max_shift/shift_step+1.0); // Number of shifts to try.
            
    for (int shiftidx=0; shiftidx<n_shifts; shiftidx++){
        float shift=-max_shift+shiftidx*shift_step;

        //shift_phases=exp(-2*pi*sqrt(-1).*rk2.*shift./(2*rmax+1)); 
        float2 c0 = ComplexExp(make_float2(0,-2.0*PI_F*(-n_r)*shift/(2*n_r+1)));
        float2 c1 = ComplexExp(make_float2(0,-2.0*PI_F*shift/(2*n_r+1)));

    // C1=2*real(P1_shifted'*P2);
    // Essentially implemeting  
    //        2.*real(ComplexMatMulConj(P1, P2, C2tmp, n_theta,n_theta,n_r));
    // without allocating an addition array        

        for (int i = 0; i < n_theta; ++i){
            for (int j = 0; j < n_theta; ++j){
                        
                corr = 0.0;
                float2 phi = c0;
                for (int k = 0; k < n_r; ++k){
                    // We only compute the real part of the product
                    float2 P1_shifted = ComplexMul(phi,P1[k+n_r*i]);
                    ab = P1_shifted.x * P2[k+n_r*j].x + P1_shifted.y * P2[k+n_r*j].y;
                    corr += ab;
                    phi = ComplexMul(phi,c1); // Generate next shift phase
                }
                corr *= 2.0;

                // The following 6 lines replace the "if" block below.
                // They implement the same "if" block below, but without
                // an if-statement, in order to avoid warp divergence.
                // Note that I have not tested extensively if that actually 
                // helps.
                float s = copysignf(1.0,corr-maxcorr);
                float a = (1.0+s)/2.0;
                float b = (1.0-s)/2.0;
    
                maxcorr = a*corr + b*maxcorr;
                cl12 = a*i + b*cl12;
                cl21 = a*j + b*cl21;
/*
                if (corr>maxcorr){ // Current pair of lines is a better candidate for the common line
                        maxcorr = corr;
                        cl12 = (float) i;
                        cl21 = (float) j;
                }
*/
            }
        }

    //  C2=2*real(P1_shifted_flipped'*P2);
    // Essentially implemeting  
    //        2.*real(ComplexMatMulTrans(P1, P2, C2tmp, n_theta,n_theta,n_r));
    // without allocating an addition array        
        for (int i = 0; i < n_theta; ++i){
            for (int j = 0; j < n_theta; ++j){
                corr = 0.0;
                float2 phi = c0;
                for (int k = 0; k < n_r; ++k){
                    // We only compute the real part of the product
                    float2 P1_shifted = ComplexMulConj(phi,P1[k+n_r*i]);
                    ab = P1_shifted.x * P2[k+n_r*j].x - P1_shifted.y * P2[k+n_r*j].y;
                    corr += ab;
                    phi = ComplexMul(phi,c1); // Generate next shift phase
                }
                corr *= 2.0;

                float s = copysignf(1.0,corr-maxcorr);
                float a = (1.0+s)/2.0;
                float b = (1.0-s)/2.0;
    
                maxcorr = a*corr + b*maxcorr;
                cl12 = a*i + b*cl12;
                cl21 = a*(j+n_theta) + b*cl21;

/*
                if (corr>maxcorr){ // Current pair of lines is a better candidate for the common line
                        maxcorr = corr;
                        cl12 = (float) i;
                        cl21 = (float) j+n_theta;
                }
*/
            }
        }        
    }
    // Return results
    clstack[k1+k2*n_projs] = cl12 + 1.0;  // Host indexing is 1-based.
    clstack[k2+k1*n_projs] = cl21 + 1.0;
    corrstack[k1+k2*n_projs]  = maxcorr;  // Writing to global memory is very slow. So return only what is necessary.
}
    