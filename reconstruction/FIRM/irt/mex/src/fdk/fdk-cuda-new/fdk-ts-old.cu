#include "hip/hip_runtime.h"
// fdk-ts.cu
// Feldkamp aka FDK backprojection for arc/flat detector.
// For detector index (t,s).
// CUDA/GPU version
// Copyright 2005-6-27, Jeff Fessler, University of Michigan

#include "jf-cuda.h"
#include "def,fdk.h"
#include "fdk-gpu.h"




//
// fdk_ts_back1_kernel()
// The FDK backprojection is *added* to the image, so the user must zero it!
//
static
#ifdef fdk_gpu
__global__
#endif
void fdk_ts_back1_kernel(
#ifndef fdk_gpu
cint ix,
cint iy,
#endif
float *image, // [nz nx ny] <- trick!
int nx,
int ny,
int nz,
float dx, // voxel size
float dy, // can be negative to cause flip
float dz,
float offset_x, // image volume center offset in pixels (usually 0)
float offset_y,
float offset_z,
cbyte *mask2, // [nx ny] 2D support mask: 0, 1, ..., nthread
byte mask_id, // 1 ... nthread
float dso, // distance from source to isocenter
float dsd, // distance from source to detector
truf is_arc,
int ns, // projection view dimensions
int nt,
float ds, // horizontal ray spacing (view sample spacing)
float dt, // vertical ray spacing (view sample spacing)
float offset_s, // channel offset [pixels]
float offset_t, // vertical offset on detector [pixels]
cfloat *proj, // [nt ns] <- trick! projection view at angle beta
float beta) // source angle [radians]
{
#ifdef fdk_gpu
	// index into image array
	cint ix = blockIdx.x * blockDim.x + threadIdx.x;
	cint iy = blockIdx.y * blockDim.y + threadIdx.y;
	
#endif

	if (ix >= nx || iy >= ny)
		return;
	
	image += (ix + iy * nx) * nz;;

	if (mask2[ix + iy*nx] != mask_id) // each thread does its part only
		return;

#ifdef fdk_gpu
	__shared__ float shared_img[2][2][120];
	float *temp_img = shared_img[threadIdx.x][threadIdx.y];
	
	//extern __shared__ float shared_img[];
	//float *temp_img = shared_img + (threadIdx.y + (threadIdx.x)*5)*nz;
	
	
	#pragma unroll
	for(int i=0; i < nz; i++){
		temp_img[i] = image[i];
	}
	
#endif


	cfloat wx = (nx-1)/2. + offset_x;
	cfloat wy = (ny-1)/2. + offset_y;
	cfloat wz = (nz-1)/2. + offset_z;
	cfloat ws = (ns-1)/2. + offset_s;
	cfloat wt = (nt-1)/2. + offset_t;
	cfloat sinb = sinf(beta);
	cfloat cosb = cosf(beta);

	cfloat yy = dy * (iy - wy);
	cfloat xx = dx * (ix - wx);
	cfloat xbeta = xx * cosb + yy * sinb;
	cfloat ybetas = dso - (-xx * sinb + yy * cosb);
	cfloat mag = dsd / ybetas;
	cfloat ss = is_arc ? (dsd * atan2f(xbeta, ybetas))
				: (mag * xbeta);
	cfloat ss_bin = ss / ds + ws;
	cint is = floorf(ss_bin); // index of nearest neighbor in "s"

	if (is < 0 || is >= ns-1) // out of FOV
		return;

	cfloat w2 = is_arc ? // fan-beam image domain weighting
		(Sqr(dsd) / (Sqr(ybetas) + Sqr(xbeta))) : Sqr(mag);

	cfloat wr = ss_bin - is; // horizontal bilinear
	cfloat wl = 1. - wr; // interpolation factors
	
#ifdef fdk_gpu
	float *pi = temp_img;
#else
	float *pi = image;
#endif
	
	float *pt = image;
	cfloat *pp1 = proj + is * nt;
	cfloat *pp2 = proj + (is+1) * nt;
	
	#pragma unroll 120
	for (int iz = 0; iz < nz; ++iz, ++pi, ++pt) { // slice loop
		cfloat zz = dz * (iz - wz);
		cfloat tt = mag * zz;
		cfloat tt_bin = tt / dt + wt;
		cint it = floorf(tt_bin); // nearest nbr in "t"

		if (it < 0 || it >= nt-1) // out of FOV
			continue;
		else {
			cfloat wu = tt_bin - it;
			cfloat wd = 1. - wu;
			cfloat p1 = wl * pp1[it]
				+ wr * pp2[it]; // interpolate
			cfloat p2 = wl * pp1[it+1]
				+ wr * pp2[it+1]; // horizontal

			// final vertical interpolation:
			//*pi += w2 * (wu * p1 + wd * p2);
			*pt = *pi + w2 * (wu * p1 + wd * p2);
			//*pt += + w2 * (wu * p1 + wd * p2);
		}
	}

/*
#ifdef fdk_gpu
	__syncthreads();
	#pragma unroll
	for(int i=0; i<nz; i++){
		image[i] = temp_img[i];
	}
#endif
*/
	
}


#ifdef fdk_gpu
static int iDivUp(int a, int b) {
	return (a % b != 0) ? (a / b + 1) : (a / b);
}
#endif

//
// fdk_ts_back1()
// The FDK backprojection is *added* to the image, so the user must zero it!
//
sof fdk_ts_back1_gpu(
float *image, // [nz nx ny] <- trick!
cint nx,
cint ny,
cint nz,
cfloat dx, // voxel size
cfloat dy, // can be negative to cause flip
cfloat dz,
cfloat offset_x, // image volume center offset in pixels (usually 0)
cfloat offset_y,
cfloat offset_z,
cbyte *mask2, // [nx ny] 2D support mask: 0, 1, ..., nthread
cbyte mask_id, // 1 ... nthread
cfloat dso, // distance from source to isocenter
cfloat dsd, // distance from source to detector
cfloat dfs, // distance from focal point to source (0 or inf)
cint ns, // projection view dimensions
cint nt,
cfloat ds, // horizontal ray spacing (view sample spacing)
cfloat dt, // vertical ray spacing (view sample spacing)
cfloat offset_s, // channel offset [pixels]
cfloat offset_t, // vertical offset on detector [pixels]
cfloat *proj, // [nt ns] <- trick! projection view at angle beta
cfloat beta) // source angle [radians]
{
	truf is_arc = 0;
	if (dfs == 0)
		is_arc = 1;
	else if (!Isinf(dfs))
		Warn("dfs not done - junk!")

	static truf told = False;
	if (!told) {
		Note2("nx=%d ny=%d", nx, ny)
		told = True;
	}

#ifdef fdk_gpu
//	dim3 dimBlock(nx, ny);
	dim3 dimBlock(2, 2);
	dim3 dimGrid(iDivUp(nx,dimBlock.x), iDivUp(ny,dimBlock.y));
	
	//call when using internal shared memory
	fdk_ts_back1_kernel<<<dimGrid, dimBlock>>>(
	
	//call when using extern shared memory
	//fdk_ts_back1_kernel<<<dimGrid, dimBlock, 6*5*nz*sizeof(float)>>>(
#else
	for (int iy=0; iy < ny; ++iy)
	for (int ix=0; ix < nx; ++ix)
		fdk_ts_back1_kernel(ix, iy,
#endif
		image,
		nx,
		ny,
		nz,
		dx,
		dy,
		dz,
		offset_x,
		offset_y,
		offset_z,
		mask2,
		mask_id,
		dso,
		dsd,
		is_arc,
		ns,
		nt,
		ds,
		dt,
		offset_s,
		offset_t,
		proj,
		beta);
	Ok
}








